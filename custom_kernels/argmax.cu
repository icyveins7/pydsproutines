#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

/*
Assuming a 4-D array, each block gets the 3-D argmax of the last 3 dimensions.

Different flavours for different input types,
but all returned indices are in uint32.
*/

// Input type: uint32
extern "C" __global__
void multiArgmax3d_uint32(
    const unsigned int *d_x,
    const int numItems,
    const int dim1,
    const int dim2,
    const int dim3, // so d_x is expected to be (numItems * dim1 * dim2 * dim3)
    unsigned int *d_argmax, // output is expected to be (numItems * 3)
    unsigned int *d_max // length of (numItems)
){
    // allocate shared memory
    extern __shared__ double s[];

    unsigned int *s_item = (unsigned int*)s; // (blockDim) unsigned ints
    unsigned int *s_idx = (unsigned int*)&s_item[blockDim.x]; // (blockDim) unsigned ints

    // First we zero both shared mem workspaces
    s_item[threadIdx.x] = 0;
    s_idx[threadIdx.x] = 0;
    // no need to sync here, each thread goes on to its own reads and comparisons first

    // extract the item for this block
    int itemSize = dim1 * dim2 * dim3;
    unsigned int item;
    for (int t = threadIdx.x; t < itemSize; t += blockDim.x)
    {
        item = d_x[itemSize * blockIdx.x + t];
        if (item > s_item[threadIdx.x])
        {
            // replace the value, and write the index
            s_item[threadIdx.x] = item;
            s_idx[threadIdx.x] = t;
        }
    }
    __syncthreads();

    // Parallel reduction maximum
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            // retain the larger value
            if (s_item[threadIdx.x] < s_item[threadIdx.x + s])
            {
                s_item[threadIdx.x] = s_item[threadIdx.x + s];
                s_idx[threadIdx.x] = s_idx[threadIdx.x + s]; // remember to take the index too
            }
        }

        __syncthreads();
    }

    // Now the max value and index is at the zero index in the workspaces
    unsigned int rem = s_idx[0];
    unsigned int out1 = rem / (dim2 * dim3);
    rem = rem % (dim2 * dim3);
    unsigned int out2 = rem / (dim3);
    rem = rem % dim3;
    unsigned int out3 = rem;
    
    // Execute 3-D write with just the first 3 threads
    if (threadIdx.x == 0)
        d_argmax[blockIdx.x * 3 + 0] = out1;
    if (threadIdx.x == 1)
        d_argmax[blockIdx.x * 3 + 1] = out2;
    if (threadIdx.x == 2)
        d_argmax[blockIdx.x * 3 + 2] = out3;
    if (d_max != NULL && threadIdx.x == 3)
        d_max[blockIdx.x] = s_item[0];

}


// Multi argmax on every row of a complex 2D matrix, with internal abs performed
// Each block tackles 1 row.
extern "C" __global__
void multiArgmaxAbsRows_complex64(
    const complex<float> *d_x,
    const int numRows, // dimension 1 of d_x
    const int length,  // dimension 2 of d_x
    unsigned int *d_argmax // output, has length numRows
){
    // allocate shared memory
    extern __shared__ double s[];

    float *s_ws = (float*)s; // (blockDim) floats
    unsigned int *s_idx = (unsigned int*)&s_ws[blockDim.x]; // (blockDim) unsigned ints

    // pre-zero the workspace
    s_ws[threadIdx.x] = 0.0f;
    s_idx[threadIdx.x] = 0;

    // define the row we are working on for this block
    const complex<float>* d_row = &d_x[blockIdx.x * length];

    // load and compare at the same time
    float absval;
    for (int t = threadIdx.x; t < length; t += blockDim.x)
    {
        absval = abs(d_row[t]);
        if (absval > s_ws[threadIdx.x]) // update the workspace values for this thread
        {
            s_ws[threadIdx.x] = absval;
            s_idx[threadIdx.x] = t;
        }
    }
    __syncthreads();

    // now compare across the workspace with parallel reductions
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s){
            if (s_ws[threadIdx.x + s] > s_ws[threadIdx.x]){
                s_ws[threadIdx.x] = s_ws[threadIdx.x + s];
                s_idx[threadIdx.x] = s_idx[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // max argument is in the first workspace value now, write it to global output
    if (threadIdx.x == 0)
        d_argmax[blockIdx.x] = s_idx[0];
}