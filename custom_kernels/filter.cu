#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

// This kernel only loads the filter taps into shared memory, without an extra workspace
// This is useful if the number of filter taps is very long, and hence leaves less than 2*length of complex workspace for the data to sit in.
// Relevant only for real-only filter taps.
// In order to not waste warps, it is recommended to set outputPerBlk to a multiple of blockDim.
extern "C" __global__
void filter_smtaps(
    const complex<float> *d_x, const int len,
    const float *d_taps, const int tapslen,
    const int outputPerBlk,
    complex<float> *d_out, int outlen,
    const complex<float> *d_delay, const int delaylen,
    const int dsr, const int dsPhase)
{
    // allocate shared memory
    extern __shared__ double s[];
    
    float *s_taps = (float*)s; // (tapslen) floats
    /* Tally:  */

    // load shared memory
    for (int t = threadIdx.x; t < tapslen; t = t + blockDim.x){
        s_taps[t] = d_taps[t];
    }
    
    __syncthreads();
    
    // Begin computations
    int i; // output index
    int k; // reference index (not equal to output if downsample is >1)
    complex<float> z; // stack-var for each thread
    for (int t = threadIdx.x; t < outputPerBlk; t = t + blockDim.x)
    {
        z = 0; // reset before the output

        i = blockIdx.x * outputPerBlk + t; // This is the thread's output index
        k = i * dsr + dsPhase; // This is the reference index

        // Exit if we hit the end
        if (i >= outlen)
            break;

        // Otherwise loop over the taps
        for (int j = 0; j < tapslen; j++)
        {
            int xIdx = k - j;

            // accumulate
            if (xIdx >= 0 && xIdx < len)
                z = z + d_x[xIdx] * s_taps[j]; // this uses the input data
            else if (delaylen + xIdx >= 0 && d_delay != NULL) // d_delay must be supplied for this to work
                z = z + d_delay[delaylen + xIdx] * s_taps[j]; // this uses the delay data (from previous invocations)
        }

        // Coalesced writes
        d_out[i] = z;
    }
 
}


// ================
// If the number of taps is small, we can allocate a workspace for the complex-valued inputs
// and then use that workspace to prevent repeated global reads of the same element
extern "C" __global__
void filter_smtaps_sminput(
    const complex<float> *d_x, const int len,
    const float *d_taps, const int tapslen,
    const int outputPerBlk,
    const int workspaceSize, // this must correspond to outputPerBlk + tapslen - 1
    complex<float> *d_out, int outlen)
{
    // allocate shared memory
    extern __shared__ double s[];
    
    float *s_taps = (float*)s; // (tapslen) floats
    complex<float> *s_ws = (complex<float>*)&s_taps[tapslen]; // workspaceSize
    /* Tally:  */

    // load shared memory taps
    for (int t = threadIdx.x; t < tapslen; t = t + blockDim.x){
        s_taps[t] = d_taps[t];
    }
    // load the shared memory workspace
    int i0 = blockIdx.x * outputPerBlk; // this is the first output index
    int workspaceStart = i0 - tapslen + 1; // this is the first index that is required
    // int workspaceEnd   = i0 + outputPerBlk; // this is the last index that is required (non-inclusive)
    int i;
    for (int t = threadIdx.x; t < workspaceSize; t = t + blockDim.x)
    {
        i = workspaceStart + t; // this is the input source index to copy
        if (i < 0 || i >= outlen) // set to 0 if its out of range
            s_ws[t] = 0;
        else
            s_ws[t] = d_x[i];
    }
    
    __syncthreads();
    
    // Begin computations
    complex<float> z; // stack-var for each thread
    int wsi;
    for (int t = threadIdx.x; t < outputPerBlk; t = t + blockDim.x)
    {
        z = 0; // reset before the output

        i = blockIdx.x * outputPerBlk + t; // This is the output index
        wsi = tapslen - 1 + t; // this is the 'equivalent' source index from shared memory

        // Exit if we hit the end
        if (i >= outlen)
            break;

        // Otherwise loop over the taps and the shared mem workspace
        for (int j = 0; j < tapslen; j++)
        {
            // accumulate
            z = z + s_ws[wsi - j] * s_taps[j];
        }

        // Coalesced writes
        d_out[i] = z;
    }
 
}