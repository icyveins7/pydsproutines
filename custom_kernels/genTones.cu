#include "hip/hip_runtime.h"
/*
Using nvprof to examine the kernel times shows that the kernels are actually slower as compared to 
the natural cupy exp(...) calls. However, the cupy calls are split into several steps, with significant 
downtime between each kernel call, which result in a slower process when the size is relatively small.
*/

#include <cupy/complex.cuh>

// this is the naive implementation, we just calculate the exact exponential for each sample
extern "C" __global__
void genTonesDirect_64f(
	const double f0,
	const double fstep,
	const int numFreqs,
	const int len,
	complex<double> *out)
{
	// spawn as many blocks as required to fulfill the length
	// each block just writes to its own section of the output, iterates over every frequency
	int startidx = blockIdx.x * blockDim.x;
	int i = threadIdx.x + startidx; // each thread will work on this sample, for every frequency
	int offset;
	
	double f, phase, re, im;
	
	for (int fidx = 0; fidx < numFreqs; fidx++)
	{
		f = f0 + fidx * fstep; // the frequency we are working on
		offset = fidx * len; // offset to the row to write to
		sincospi(2 * f * (double)i, &im, &re); // write the components to stack
		if (i < len){ // coalesced global writes
			out[offset + i] = complex<double>(re, im);
		}
	}
}

// this doesn't calculate for each frequency, but just multiplies the original values by a constant complex number
// there might be some loss due to computational error build up over many frequencies
// note that even though python timing may show this to be slower, the kernel timing for this is around 10x faster than the direct one
extern "C" __global__
void genTonesScaling_64f(
	const double f0,
	const double fstep,
	const int numFreqs,
	const int len,
	complex<double> *out)
{
	// spawn as many blocks as required to fulfill the length
	// each block just writes to its own section of the output, iterates over every frequency
	int startidx = blockIdx.x * blockDim.x;
	int i = threadIdx.x + startidx; // each thread will work on this sample, for every frequency
	int offset = 0;
	
	double f, phase, re, im;
	complex<double> outstack;
	
	// calculate the value for the first frequency
	sincospi(2 * f0 * (double)i, &im, &re);
	outstack = complex<double>(re, im); // we keep a copy on stack
	if (i < len){ // coalesced global writes
		out[offset + i] = outstack;
	}
	
	// now calculate the complex number to scale by
	sincospi(2 * fstep * (double)i, &im, &re);
	complex<double> alpha = complex<double>(re, im);
	
	// loop over the rest of the frequencies
	for (int fidx = 1; fidx < numFreqs; fidx++) // now start from 1
	{
		f = f0 + fidx * fstep; // the frequency we are working on
		offset = fidx * len; // offset to the row to write to
		outstack = outstack * alpha;
		if (i < len){
			out[offset + i] = outstack;
		}
	}
}

// ============================== 32f versions ======================================================
// note that internally it's still computed with doubles, but we write to global mem as floats
// as such, the extra explicit casts actually make it slightly slower than the 64f version
extern "C" __global__
void genTonesDirect_32f(
	const double f0,
	const double fstep,
	const int numFreqs,
	const int len,
	complex<float> *out)
{
	// spawn as many blocks as required to fulfill the length
	// each block just writes to its own section of the output, iterates over every frequency
	int startidx = blockIdx.x * blockDim.x;
	int i = threadIdx.x + startidx; // each thread will work on this sample, for every frequency
	int offset;
	
	double f, phase;
    double re, im;
	
	for (int fidx = 0; fidx < numFreqs; fidx++)
	{
		f = f0 + fidx * fstep; // the frequency we are working on
		offset = fidx * len; // offset to the row to write to
		sincospi(2 * f * (double)i, &im, &re); // write the components to stack
		if (i < len){ // coalesced global writes
			out[offset + i] = complex<float>(re, im);
		}
	}
}

// again, computed with doubles, saved as floats
extern "C" __global__
void genTonesScaling_32f(
	const double f0,
	const double fstep,
	const int numFreqs,
	const int len,
	complex<float> *out)
{
	// spawn as many blocks as required to fulfill the length
	// each block just writes to its own section of the output, iterates over every frequency
	int startidx = blockIdx.x * blockDim.x;
	int i = threadIdx.x + startidx; // each thread will work on this sample, for every frequency
	int offset = 0;
	
	double f, phase, re, im;
	complex<double> outstack;
	
	// calculate the value for the first frequency
	sincospi(2 * f0 * (double)i, &im, &re);
	outstack = complex<double>(re, im); // we keep a copy on stack
	if (i < len){ // coalesced global writes
		out[offset + i] = outstack;
	}
	
	// now calculate the complex number to scale by
	sincospi(2 * fstep * (double)i, &im, &re);
	complex<double> alpha = complex<double>(re, im);
	
	// loop over the rest of the frequencies
	for (int fidx = 1; fidx < numFreqs; fidx++) // now start from 1
	{
		f = f0 + fidx * fstep; // the frequency we are working on
		offset = fidx * len; // offset to the row to write to
		outstack = outstack * alpha; 
		if (i < len){
			out[offset + i] = complex<float>(outstack.real(), outstack.imag());
		}
	}
}

/* 
What about dot producting tones directly? 
This would be like CZTs, except possibly without the overhead of multiple FFT calls and optimised shared mem usage?
Benefit of this is to read the source only once from global memory.
*/

// 1. We use shared memory to store up to 64 * 64 complex64 values of src*tone. To keep things simple, we can fix this size, along with the kernel parameters, as 64x64 always.
// 2. Then we sum up within shared memory and then output to an external array (this will then require a separate second kernel to sum results together)
// 3. Return to step 1 with the rest of the frequencies, until all frequencies are complete.
extern "C" __global__
void dotTonesScaling_32f(
	const double f0,
	const double fstep,
	const int numFreqs,
	const int len,
	const complex<float> *src,
	complex<float> *out)
{
   // spawn as many blocks as required to fulfill the length
	// each block just writes to its own section of the output, iterates over every frequency
	int startidx = blockIdx.x * blockDim.x;
	int i = threadIdx.x + startidx; // each thread will work on this sample, for every frequency

   // initialise shared memory
   extern __shared__ double s[];   
   complex<float> *s_ws = (complex<float>*)s; // (64*64) complex floats
   
   // let's have a variable to mark the row in shared memory we are currently writing to
   int s_row;

   // the rest of the variable declarations as before
	double f, phase, re, im;
	complex<double> outstack;
	
	// calculate the value for the first frequency
	sincospi(2 * f0 * (double)i, &im, &re);
	outstack = complex<double>(re, im); // we keep a copy on stack
	
	// now calculate the complex number to scale by
	sincospi(2 * fstep * (double)i, &im, &re);
	complex<double> alpha = complex<double>(re, im);
    
    // make a double version of the global mem source
    complex<double> src64f;
    
    // track the batch we're on
    int batch = 0;
    // and an accumulator within batches
    complex<float> accumulator;
    // and also resolve the number of shared mem columns we actually use out of the 64
    int kMax = len - startidx < 64 ? len - startidx : 64;
	
	// no point doing anything if this thread is beyond the length
	if (i < len)
	{
    	// loop over all the frequencies
    	for (int fidx = 0; fidx < numFreqs; fidx++)
    	{
            // finally, if we have completed a batch of 64 (or we're on the last freq),
            // it's time to sum up in sharedmem and output to global mem after
            if (fidx % 64 == 0 & fidx != 0)
            {
                // we wait for everyone to be done writing to shared mem for this batch
                __syncthreads();
                
                // then we iterate across the columns
                accumulator = 0.0f; // pre-zero-ing
                for (int k = 0; k < kMax; k++) // note that not all columns may be used (if we're the last block)
                {
                    // each thread works on one particular row
                    accumulator += s_ws[threadIdx.x * 64 + k];
                }
                
                // once complete, we flush to global memory
                out[blockIdx.x * numFreqs + 64*batch + threadIdx.x] = accumulator;
                
                // increment the batch so we know where to write to next time
                batch++;
            }
            
        	// define the row in shared memory we will be working on
        	s_row = fidx % 64;
        	
        	// for everything other than the first loop, we must multiply by the tone
        	if (fidx > 0)
        	{
        		outstack = outstack * alpha; 
        	}
        	else // for the first loop, we instead now multiply by the source (this is the only time we read the source)
        	{
                src64f = complex<double>(src[i].real(), src[i].imag());
            	outstack = outstack * src64f;
        	}

        	// after we have done the multiply of the tone, we save it in the appropriate spot in the workspace on shared mem
            s_ws[64 * s_row + threadIdx.x] = complex<float>(outstack.real(), outstack.imag()); // cast to floats just like before
    	}
        
        // we must also write the last batch in..
        __syncthreads();
        
        // note that on the last batch, not all 64 rows (i.e. freqs) may have been used, so we only accumulate and write those that are remaining
        if (threadIdx.x < numFreqs - batch * 64)
        {
            // then we iterate across the columns
            accumulator = 0.0f;
            for (int k = 0; k < kMax; k++)
            {
                // each thread works on one particular row
                accumulator += s_ws[threadIdx.x * 64 + k];
            }
            
            // once complete, we flush to global memory
            out[blockIdx.x * numFreqs + 64*batch + threadIdx.x] = accumulator;
        }
	}
}

