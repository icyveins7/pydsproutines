#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

/*
We design a kernel to multiply arbitrary slices from an input 1-D array with
indexed arrays from a second input 2-D matrix.

Long input array x is sliced arbitrarily at different indices:
A-B, C-D, E-F, ... where each slice is no longer than length N.

Second input array y is a 2D matrix with individual rows presenting unique
inputs.

The goal is to multiply each slice with a particular row of the matrix. This is specified
with additional index arrays; in general this can be accomplished by iterating over
each slice and multiplying by the appropriate row, at the appropriate length.

However, this kernel attempts to exploit optimistic cases where
number of rows of y <<< number of slices from x.
This means that there is a high probability for the row to be reused from slice to slice,
preventing additional global memory reads.

Each block will allocate shared memory to hold one row, and will swap it out if there is a change
in the row index for the current slice.
*/
extern "C" __global__
void multiplySlicesWithIndexedRowsOptimistic(
    const complex<float> *d_x,
    const int xlength,
    const complex<float> *d_rows,
    const int rowLength,
    const int numRows,
    const int *d_sliceStarts, // length numSlices
    const int *d_sliceLengths, // length numSlices
    const int numSlices,
    const int *d_rowIdxs, // length numSlices
    complex<float> *d_out, // numSlices * rowLength
    int outlength)
{
    // allocate shared memory
    extern __shared__ double s[];

    complex<float> *s_row = (complex<float>*)s; // (rowLength) complex floats

    // On initialization, the loaded index is set to an invalid value
    int loadedRow = -1;
    int requiredRow;

    // Allocate stack variables to hold the current slice indices
    int sliceStart, sliceLength;

    // Iterate over the slices
    for (int i = blockIdx.x; i < numSlices; i += gridDim.x) // each block computes 1 slice at a time
    {
        // First we read the required row index
        requiredRow = d_rowIdxs[i];

        // Then we update shared memory if it's required
        if (requiredRow != loadedRow)
        {
            for (int t = threadIdx.x; t < rowLength; t = t + blockDim.x)
            {
                s_row[t] = d_rows[requiredRow * rowLength + t];
            }

            // Wait for it to be fully loaded
            __syncthreads();
        }

        // Then we perform the multiplies
        sliceStart = d_sliceStarts[i];
        sliceLength = d_sliceLengths[i];

        // Write the output to global mem
        for (int t = threadIdx.x; t < sliceLength; t += blockDim.x)
        {
            if (sliceStart + t >= 0 && sliceStart + t < xlength)
                d_out[i*outlength + t] = s_row[t] * d_x[sliceStart + t];    
        }
            
        
    }

 
}


/*
Here we design a sliding template multiply, with an additional sliding norm calculation.
This is usually used in the xcorr step.

We have two inputs:
1) x: A short (can be contained in shared mem) template array, length xlen
2) y: Another arbitrarily long input array to slide against

This is the MOST OPTIMISTIC method for the copies;
we assume that within the shared memory we can fit both
1) x itself
2) a large section of y

This allows us to slide against multiple xlen windows within 1 block,
escaping a lot of repeated global memory reads.

This is especially important when the template itself is very short;
as the template gets longer this matters less and less when compared to the next 
step in the xcorr, which is the FFT step.

*/
extern "C" __global__ 
void slidingMultiply(
    const complex<float> *x, // the template
    const int xlen,
    const complex<float> *y, // the searched array
    const int ylen,
    const int startIdx, // the start index of the searched array to begin the sliding
    const int idxlen, // the total number of slides
    complex<float> *z, // the output array, which has dimensions (idxlen) rows * (xlen) columns
    float *ynormSq, // the norms of the slices of y, may be left as NULL if undesired, dimensions
    int numSlidesPerBlk // this defines the number of slides to compute per block, and hence determines the workspace (which the caller must calculate correctly)
){
    // allocate shared memory
    extern __shared__ double s[];

    complex<float> *s_x = (complex<float>*)s; // (xlen) complex floats
    complex<float> *s_ysection = (complex<float>*)&s_x[xlen]; // (numSlidesPerBlk + xlen - 1) complex floats
    float *s_ynormSq = (float*)&s_ysection[numSlidesPerBlk + xlen - 1]; // (numSlidesPerBlk) floats

    // Load shared mem x and y
    for (int t = threadIdx.x; t < xlen; t += blockDim.x)
        s_x[t] = x[t];

    int ysectionSize = numSlidesPerBlk + xlen - 1;
    int ysectionOffset = blockIdx.x * numSlidesPerBlk;
    for (int t = threadIdx.x; t < ysectionSize; t += blockDim.x)
        s_ysection[t] = y[ysectionOffset + t];

    // Zero the shared mem norm squared if output is desired
    if (ynormSq != NULL)
    {
        for (int t = threadIdx.x; t < numSlidesPerBlk; t += blockDim.x)
            s_ynormSq[t] = 0.0f;
    }

    // Begin the sliding multiplies
    for (int i = 0; i < numSlidesPerBlk; i++)
    {
        for (int t = threadIdx.x;)
    }
    for (int t = threadIdx.x; t < numSlidesPerBlk; t += blockDim.x)
    {
        for (int i = 0; i < )

        // TODO
    }



}