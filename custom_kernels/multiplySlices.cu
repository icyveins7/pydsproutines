#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

/*
We design a kernel to multiply arbitrary slices from an input 1-D array with
indexed arrays from a second input 2-D matrix.

Long input array x is sliced arbitrarily at different indices:
A-B, C-D, E-F, ... where each slice is no longer than length N.

Second input array y is a 2D matrix with individual rows presenting unique
inputs.

The goal is to multiply each slice with a particular row of the matrix. This is specified
with additional index arrays; in general this can be accomplished by iterating over
each slice and multiplying by the appropriate row, at the appropriate length.

However, this kernel attempts to exploit optimistic cases where
number of rows of y <<< number of slices from x.
This means that there is a high probability for the row to be reused from slice to slice,
preventing additional global memory reads.

Each block will allocate shared memory to hold one row, and will swap it out if there is a change
in the row index for the current slice.
*/
extern "C" __global__
void multiplySlicesWithIndexedRowsOptimistic(
    const complex<float> *d_x,
    const int xlength,
    const complex<float> *d_rows,
    const int rowLength,
    const int numRows,
    const int *d_sliceStarts, // length numSlices
    const int *d_sliceLengths, // length numSlices
    const int numSlices,
    const int *d_rowIdxs, // length numSlices
    complex<float> *d_out, // numSlices * rowLength
    int outlength)
{
    // allocate shared memory
    extern __shared__ double s[];

    complex<float> *s_row = (complex<float>*)s; // (rowLength) complex floats

    // On initialization, the loaded index is set to an invalid value
    int loadedRow = -1;
    int requiredRow;

    // Allocate stack variables to hold the current slice indices
    int sliceStart, sliceLength;

    // Iterate over the slices
    for (int i = blockIdx.x; i < numSlices; i += gridDim.x) // each block computes 1 slice at a time
    {
        // First we read the required row index
        requiredRow = d_rowIdxs[i];

        // Then we update shared memory if it's required
        if (requiredRow != loadedRow)
        {
            for (int t = threadIdx.x; t < rowLength; t = t + blockDim.x)
            {
                s_row[t] = d_rows[requiredRow * rowLength + t];
            }

            // Wait for it to be fully loaded
            __syncthreads();
        }

        // Then we perform the multiplies
        sliceStart = d_sliceStarts[i];
        sliceLength = d_sliceLengths[i];

        // Write the output to global mem
        for (int t = threadIdx.x; t < sliceLength; t += blockDim.x)
            d_out[i*outlength + t] = s_row[t] * d_x[sliceStart + t];
        
    }

 
}