#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

// Note that this kernel does not store any of the input array into shared memory.
// It is expected that sufficient blocks are spawned to cover the length of the input.
// See the other version for a one block-one input kernel.
extern "C" __global__
void upfirdn_naive(
    const complex<float> *d_x, const int len,
    const float *d_taps, const int tapslen,
    const int up,
    const int down,
    complex<float> *d_out,
    int outlen,
    float *d_outabs)
{
    // allocate shared memory
    extern __shared__ double s[];
    
    float *s_taps = (float*)s; // (tapslen) floats
    /* Tally:  */

    // load shared memory
    for (int t = threadIdx.x; t < tapslen; t = t + blockDim.x){
        s_taps[t] = d_taps[t];
    }

    __syncthreads();
    
    // Define the index that each thread will work on (the output index)
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = i * down; // j will point to the index of the upsampled interrim
    int k; // we use this to help us point to the original source, depending on the filter tap iteration

    complex<float> z = 0; // stack-variable for the thread's output

    // only compute if we are within range
    if (i < outlen)
    {
        // we loop over the filter taps
        for (int f = 0; f < tapslen; f++)
        {
            k = j - f;
            // again, only accumulate if we are within range, and we are at a non-zero value of the upsample
            if ((k % up == 0) && (k / up < len) && (k / up >= 0))
            {
                z += s_taps[f] * d_x[k / up];
            }
        }

        // write the output to global memory
        d_out[i] = z;

        if (d_outabs != NULL)
        {
            d_outabs[i] = abs(z);
        }
    }
 
}

// This kernel attempts to compute upfirdn for one signal in each block.
// Hence spawn as many blocks as there are signals.
// TODO: TEST

extern "C" __global__
void upfirdn_sm(
    const complex<float> *d_x, const int len,
    const float *d_taps, const int tapslen,
    const int up,
    const int down,
    complex<float> *d_out,
    int outlen,
    float *d_outabs)
{
    // Calculate the required input length for the workspace, including filter lookback
    const int interrimLength = ((blockDim.x-1) * down + tapslen);
    const int inputWorkspaceLength = interrimLength % up == 0 ? interrimLength / up : interrimLength / up + 1;

    // allocate shared memory
    extern __shared__ double s[];
    
    float *s_taps = (float*)s; // (tapslen) floats
    complex<float> *s_xws = (complex<float>*)&s_taps[tapslen]; // (inputWorkspaceLength) complex floats

    // load taps
    for (int t = threadIdx.x; t < tapslen; t = t + blockDim.x){
        s_taps[t] = d_taps[t];
    }
    // zero the input workspace
    for (int t = threadIdx.x; t < inputWorkspaceLength; t = t + blockDim.x){
        s_xws[t] = 0;
    }
    __syncthreads();

    // Loop over the block until we cover the entire input
    const complex<float> *d_row = &d_x[blockIdx.x * len];
    int l0, n0, l, m, n, lws;
    complex<float> out;
    int numLoopsRequired = len % blockDim.x == 0 ? len / blockDim.x : len / blockDim.x + 1;
    for (int i = 0; i < numLoopsRequired; i++)
    {
        // Determine the first output index
        n0 = i * blockDim.x;

        // Determine the first input index required
        l0 = (n0 * down - (tapslen-1)) % up == 0 ? (n0 * down - (tapslen-1)) / up : (n0 * down - (tapslen-1)) / up + 1;

        // Copy the input workspace
        for (int t = threadIdx.x; t < inputWorkspaceLength; t = t + blockDim.x){
            // Note that for the first loop, this may be negative, so don't read out of range
            if (l0 + t >= 0)
                s_xws[t] = d_row[l0 + t];
        }
        __syncthreads();

        // Define the global output index for this thread
        n = threadIdx.x + n0;

        // Perform the accumulation
        out = 0;
        for (int f = 0; f < tapslen; f++)
        {
            // What is the interrim index?
            m = n * down - f;

            // Does this correspond to an input index?
            if (m % up == 0)
            {
                // Then what is the global input index?
                l = m / up;
                // What is its associated workspace index?
                lws = l - l0;

                // Accumulate the product
                out += s_taps[f] * s_xws[lws];
            }       
        }

        // Write to global output
        d_out[blockIdx.x * len + n] = out;
        if (d_outabs != NULL)
            d_outabs[blockIdx.x * len + n] = abs(out);
    }
 
}
